#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
/* #include <criterion/criterion.h> */
#include <hip/driver_types.h>
/* #include <hip/hip_runtime.h> */
#include "include/test_cuda_api.h"

/* void setup(void) { */
/*     puts("Runs before the test"); */
/* } */
/*  */
/* void teardown(void) { */
/*     puts("Runs after the test"); */
/* } */

__global__ void compute_pixel_value(unsigned char* image, float* pixel_value, float* min_max, int x_size, int y_size)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    //To compute laplacian of a pixel, it need 8 neighbors -> need to check range.
    if( (idx>0 && idx<x_size-1) && (idy > 0 && idy < y_size-1) )
    {
        int index = idx + idy*x_size; // current pixel for this thread
        int i,j; 
        int weight[3][3] = {{ 1, 1, 1 }, { 1, -8, 1 }, { 1, 1, 1 }};
        for (j = - 1; j < 2; j++) 
        {
            for (i = -1; i < 2; i++) 
            {
                int index_t = (idy+j)*x_size + idx + i; 
                pixel_value[index] += weight[j + 1][i + 1] * image[index_t];
            }
        }
        if (pixel_value[index] < min_max[0]) min_max[0] = pixel_value[index]; // min = min_max[0]
        if (pixel_value[index] > min_max[1]) min_max[1] = pixel_value[index];
    }
}

/* Test(simple, test, .init = setup, .fini = teardown) { */
int main(void)
{

    struct hipFuncAttributes attr;
    /* const char* fname = "compute_pixel_value"; */
    /* int nDevices; */
    /* hipGetDeviceCount(&nDevices); */
    /* for (int i = 0; i < nDevices; i++) { */
    /*     hipDeviceProp_t prop; */
    /*     hipGetDeviceProperties(&prop, i); */
    /*     printf("Device Number: %d\n", i); */
    /*     printf("  Device name: %s\n", prop.name); */
    /*     printf("  Memory Clock Rate (KHz): %d\n", */
    /*             prop.memoryClockRate); */
    /*     printf("  Memory Bus Width (bits): %d\n", */
    /*             prop.memoryBusWidth); */
    /*     printf("  Peak Memory Bandwidth (GB/s): %f\n\n", */
    /*             2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6); */
    /* } */

    memset(&attr, 0, sizeof(struct hipFuncAttributes));
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((void*))compute_pixel_value);
    /* dump_cudaFuncAttributes(attr); */

    printf("binaryVersion: %d\n", attr.binaryVersion);
    printf("cacheModeCA: %d\n", attr.cacheModeCA);
    printf("constSizeBytes: %lu\n", attr.constSizeBytes);
    return 0;

    /* cr_assert(attr.binaryVersion >= 32760); */
    /* cr_assert(attr.cacheModeCA == 0); */
    /* cr_assert(attr.constSizeBytes == 0); */
}
