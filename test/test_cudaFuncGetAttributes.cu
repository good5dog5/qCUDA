#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/driver_types.h>
#include <criterion/criterion.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void dump_cudaFuncAttributes(struct hipFuncAttributes attr)
{
    printf("binaryVersion: %d\n", attr.binaryVersion);
    printf("cacheModeCA: %d\n", attr.cacheModeCA);
    printf("constSizeBytes: %lu\n", attr.constSizeBytes);
    printf("localSizeBytes: %lu\n", attr.localSizeBytes);
    printf("maxThreadPerBlock: %d\n", attr.maxThreadsPerBlock);
    printf("numRegs: %d\n", attr.numRegs);
    printf("sharedSizeBytes: %lu\n", attr.sharedSizeBytes);
}
void setup(void) {
    puts("Runs before the test");

}

void teardown(void) {
    puts("Runs after the test");
}

/* Test(FuncGetAttribute, first) { */
/*     struct hipFuncAttributes attr; */
/*     hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(saxpy)); */
/*     cr_assert(attr.binaryVersion >= 32760); */
/*     cr_assert(attr.cacheModeCA >= 0); */
/*     cr_assert(attr.constSizeBytes >= 0); */
/*      */
/* } */
int main(void)
{

    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(saxpy));
    dump_cudaFuncAttributes(attr);
    assert(attr.binaryVersion >= 32760);
    assert(attr.cacheModeCA >= 0);
    assert(attr.constSizeBytes >= 0);

}
