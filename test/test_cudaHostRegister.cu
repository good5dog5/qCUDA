#include <stdio.h>
#include <hip/hip_runtime.h>
#define SIZE 10

// Kernel definition, see also section 4.2.3 of Nvidia Cuda Programming Guide

__global__ void vecAdd(float* A, float* B, float* C) {
    // threadIdx.x is a built-in variable provided by CUDA at runtime
    int i = threadIdx.x;
    //includeA[i] = 0;
    //includeB[i] = i;
    C[i] = A[i] + B[i];
    printf("Kernel: A[%d]=%f, B[%d]=%f, C[%d]=%f\n", i, A[i], i, B[i], i, C[i]);
}

int main() {
    int N = SIZE;

//includeround up the size of the array to be a multiple of the page size
    size_t memsize = ((SIZE * sizeof(float) + 4095) / 4096) * 4096;
    hipDeviceProp_t deviceProp;
    // Get properties and verify device 0 supports mapped memory
    hipGetDeviceProperties(&deviceProp, 0);
    if (!deviceProp.canMapHostMemory) {
            fprintf(stderr, "Device %d cannot map host memory!\n", 0);
            exit(EXIT_FAILURE);
        }

    // set the device flags for mapping host memory

    hipSetDeviceFlags(hipDeviceMapHost);
    float * A, *B, *C;
    float *devPtrA, *devPtrB, *devPtrC;
//includeuse valloc instead of malloc
    A = (float*) valloc(memsize);
    B = (float*) valloc(memsize);
    C = (float*) valloc(memsize);
    hipHostRegister(A, memsize, hipHostRegisterMapped);
    hipHostRegister(B, memsize, hipHostRegisterMapped);
    hipHostRegister(C, memsize, hipHostRegisterMapped);
    for (int i = 0; i < SIZE; i++) {
            A[i] = B[i] = i;
        }

    hipHostGetDevicePointer((void **) &devPtrA, (void *) A, 0);
    hipHostGetDevicePointer((void **) &devPtrB, (void *) B, 0);
    hipHostGetDevicePointer((void **) &devPtrC, (void *) C, 0);

    vecAdd<<<1, N>>>(devPtrA, devPtrB, devPtrC);
    hipDeviceSynchronize();

    for (int i = 0; i < SIZE; i++)
        printf("C[%d]=%f\n", i, C[i]);

    hipHostUnregister(A);
    hipHostUnregister(B);
    hipHostUnregister(C);
    free(A);
    free(B);
    free(C);

}
