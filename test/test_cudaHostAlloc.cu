#include <stdio.h>
#include <hip/hip_runtime.h>                                                                                                         

// includes                                                                                                                       
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples                                              
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization                                     

#include <hip/hip_runtime.h>    

int main()
{
    unsigned long long * p = NULL;
    unsigned long long cb = 5368709120;
    hipError_t rval;
    rval = hipHostAlloc( &p, cb, hipHostMallocPortable | hipHostMallocMapped );
    /* hipError_t rval = hipHostAlloc( &p, cb, hipHostMallocWriteCombined); */
    printf( "hipHostAlloc( ..., %llu, ... ) returns %d\n", cb, rval );
    checkCudaErrors(hipHostAlloc( &p, cb, hipHostMallocPortable | hipHostMallocMapped ));

    cb = 3098115388;
    rval = hipHostAlloc( &p, cb, hipHostMallocPortable | hipHostMallocMapped );
    printf( "hipHostAlloc( ..., %llu, ... ) returns %d\n", cb, rval );
    checkCudaErrors(hipHostAlloc( &p, cb, hipHostMallocPortable | hipHostMallocMapped ));
}
