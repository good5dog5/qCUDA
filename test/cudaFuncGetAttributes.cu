#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/driver_types.h>
#include <criterion/criterion.h>

__constant__ float constPool[100];
__global__ void saxpy(int n, float a, float *x, float *y)
{
  static a[10] = {0};
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i] + constPool[i/100];
}

void dump_cudaFuncAttributes(struct hipFuncAttributes attr)
{
    printf("binaryVersion: %d\n", attr.binaryVersion);
    printf("cacheModeCA: %d\n", attr.cacheModeCA);
    printf("constSizeBytes: %lu\n", attr.constSizeBytes);
    printf("localSizeBytes: %lu\n", attr.localSizeBytes);
    printf("maxThreadPerBlock: %d\n", attr.maxThreadsPerBlock);
    printf("numRegs: %d\n", attr.numRegs);
    printf("sharedSizeBytes: %lu\n", attr.sharedSizeBytes);
}

int main(void)
{
    struct hipFuncAttributes attr;

    int size=200;
    float darray[200];

    memset(darray, 0, sizeof(float)*200);
    hipMemcpyToSymbol(HIP_SYMBOL(constPool),  darray,   sizeof(float)*100);

    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(saxpy));
    dump_cudaFuncAttributes(attr);

    /* assert(attr.binaryVersion >= 32760); */
    /* assert(attr.cacheModeCA >= 0); */
    /* assert(attr.constSizeBytes >= 0); */

}
