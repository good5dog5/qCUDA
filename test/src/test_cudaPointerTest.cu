#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>

#define COLOR_RED     "\x1b[31m"
#define COLOR_RESET   "\x1b[0m"


unsigned int BKDRHash(char *ptr, int size)
{
    unsigned int hash = 0;
    for(int i=0; i<size; i++)
        hash = (hash<<5) - hash + (*ptr++);
    return (hash);
}
void dump_proc_maps(void)
{
  pid_t PID = getpid();
  char cmd[1000];
  sprintf(cmd, "cat /proc/%d/maps", (int)PID);
  system(cmd);
}

void count_proc_maps(void)
{
  pid_t PID = getpid();
  char buf[1000];
  FILE *f;
  sprintf(buf, "/proc/%d/maps", (int)PID);

  f = fopen(buf, "rt");

  while (fgets(buf, 1000, f)) {
      unsigned int from, to, pgoff, major, minor;
      unsigned long ino;
      char flags[4];
      char info[200] = {0};
      int ret = sscanf(buf, "%x-%x %4c %x %x:%x %lu %100c ", &from, &to, flags, &pgoff, &major, &minor, &ino, info);

      // VMA that includes mmap allocated
      if (strcmp(info, "/dev/qcuda\n") == 0) {
          
          /* printf("%s", info); */
          printf("%x-%x %4c %x %x:%x %lu %s   ", from, to, flags, pgoff, major, minor, ino, "/dev/qcuda");
          printf("Size:%7d   #pages(/4096):%5d\n", to-from, (to-from)/4096);
      }
        

      /* printf("%") */
      if (ret > 8)
          break;
  }
  printf("\n\n\n");
}

int main(int argc, char * argv[])
{

  unsigned long SIZE; 
  int freezeSec; 
  void * value;
  void **ptr2;

  SIZE      = atoi(argv[1]);
  freezeSec = atoi(argv[2]);
  ptr2      = &value;


  /* printf("Before hipHostAlloc's maps\n"); */
  /* dump_proc_maps(); */
  /* printf(COLOR_RED     "This text is RED!"     COLOR_RESET "\n"); */
  printf(COLOR_RED "Allocated %ld Kbytes" COLOR_RESET "\n", SIZE/1024);
  /* for(int i=0; i<SIZE; i++) {value[i] = 799;} */
  /* printf(COLOR_RED "[Before] hash val is %u" COLOR_RESET "\n", BKDRHash((char*)value, sizeof(int)*SIZE)); */

  hipHostAlloc(ptr2, SIZE, 0);
  for (int i =0; i<SIZE/4; i++) {
    printf("%c", *((char*)(value+i)));
  }
  printf("\n");
  printf("After hipHostAlloc's maps\n");
  count_proc_maps();
  /* dump_proc_maps(); */
  sleep(freezeSec);


  /* printf(COLOR_RED "[After] hash val is %u" COLOR_RESET "\n", BKDRHash((char*)value, sizeof(int)*SIZE)); */
  /* free(value); */
  /* for(int i=0; i<SIZE; i++) {printf("%d ", value[i]);} */
  printf("PASSED\n");
  return 0;
}
