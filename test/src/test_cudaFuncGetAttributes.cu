#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/driver_types.h>
#include <criterion/criterion.h>

__constant__ float pool[100];

__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i]+pool[i/100];
}

void dump_cudaFuncAttributes(struct hipFuncAttributes attr)
{
    printf("binaryVersion: %d\n", attr.binaryVersion);
    printf("cacheModeCA: %d\n", attr.cacheModeCA);
    printf("constSizeBytes: %lu\n", attr.constSizeBytes);
    printf("localSizeBytes: %lu\n", attr.localSizeBytes);
    printf("maxThreadPerBlock: %d\n", attr.maxThreadsPerBlock);
    printf("numRegs: %d\n", attr.numRegs);
    printf("sharedSizeBytes: %lu\n", attr.sharedSizeBytes);
}
int main(void)
{


    struct hipFuncAttributes attr;

    // for constSizeBytest (cuda const memory)
    float host[100];
    hipMemset (host,0,sizeof(float)*100);
    /* hipMemcpyToSymbol(HIP_SYMBOL(pool),  host,   sizeof(float)*100  ); */

    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(saxpy));
    dump_cudaFuncAttributes(attr);
    /* assert(attr.binaryVersion == 21); */
    /* assert(attr.cacheModeCA == 0); */
    /* assert(attr.constSizeBytes == 400); */

}
