
#include <hip/hip_runtime.h>
#include <stdio.h>
#define DSIZE (4*1024*1024)

int main(){

  int *data;
  hipFree(0);
  system("cat /proc/meminfo > out1.txt");
  printf("*$*before alloc\n");
  hipHostAlloc(&data, DSIZE, hipHostMallocDefault);
  printf("*$*after alloc\n");
  system("cat /proc/meminfo > out2.txt");
  hipHostFree(data);
  system("cat /proc/meminfo > out3.txt");
  printf("OK");
  return 0;
}
