#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/driver_types.h>
#include <criterion/criterion.h>


void dump_cudaFuncAttributes(struct hipFuncAttributes attr)
{
    printf("binaryVersion: %d\n", attr.binaryVersion);
    printf("cacheModeCA: %d\n", attr.cacheModeCA);
    printf("constSizeBytes: %lu\n", attr.constSizeBytes);
    printf("localSizeBytes: %lu\n", attr.localSizeBytes);
    printf("maxThreadPerBlock: %d\n", attr.maxThreadsPerBlock);
    printf("numRegs: %d\n", attr.numRegs);
    printf("sharedSizeBytes: %lu\n", attr.sharedSizeBytes);
}
void setup(void) {
    puts("Runs before the test");

}

void teardown(void) {
    puts("Runs after the test");
}

/* Test(FuncGetAttribute, first) { */
/*     struct hipFuncAttributes attr; */
/*     hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(saxpy)); */
/*     cr_assert(attr.binaryVersion >= 32760); */
/*     cr_assert(attr.cacheModeCA >= 0); */
/*     cr_assert(attr.constSizeBytes >= 0); */
/*      */
/* } */
__constant__ float cangle[360];

__global__ void test_kernel(float* darray)
{
    int index;

    //calculate each thread global index
    index = blockIdx.x * blockDim.x + threadIdx.x;

#pragma unroll 10
    for(int loop=0;loop<360;loop++)
        darray[index]= darray [index] + cangle [loop] ;
    return;

}


int main(int argc,char** argv)
{
    int size=3200;
    float* darray;
    float hangle[360];

    //allocate device memory
    hipMalloc ((void**)&darray,sizeof(float)*size);

    //initialize allocated memory
    hipMemset (darray,0,sizeof(float)*size);

    //initialize angle array on host
    for(int loop=0;loop<360;loop++)
        hangle[loop] = acos( -1.0f )* loop/ 180.0f;

    //copy host angle data to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(  cangle),  hangle,   sizeof(float)*360  );

    /* test_kernel  <<<  size/64  ,64  >>>  (darray); */

    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(test_kernel));
    dump_cudaFuncAttributes(attr);

    //free device memory
    hipFree(darray);
    return 0;
}


