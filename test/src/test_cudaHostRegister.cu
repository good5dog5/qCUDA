#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#define SIZE 10

// Kernel definition, see also section 4.2.3 of Nvidia Cuda Programming Guide

/* __global__ void vecAdd(float* A, float* B, float* C) { */
/*     // threadIdx.x is a built-in variable provided by CUDA at runtime */
/*     int i = threadIdx.x; */
/*     //includeA[i] = 0; */
/*     //includeB[i] = i; */
/*     C[i] = A[i] + B[i]; */
/*     printf("Kernel: A[%d]=%f, B[%d]=%f, C[%d]=%f\n", i, A[i], i, B[i], i, C[i]); */
/* } */

int main() {
    int N = SIZE;

//includeround up the size of the array to be a multiple of the page size
    size_t memsize = ((SIZE * sizeof(float) + 4095) / 4096) * 4096;
    hipDeviceProp_t deviceProp;
    // Get properties and verify device 0 supports mapped memory
    hipGetDeviceProperties(&deviceProp, 0);
    if (!deviceProp.canMapHostMemory) {
            fprintf(stderr, "Device %d cannot map host memory!\n", 0);
            exit(EXIT_FAILURE);
        }

    // set the device flags for mapping host memory

    printf("1\n");
    hipSetDeviceFlags(hipDeviceMapHost);
    printf("2\n");
    float * A;
    /* float *B, *C; */
    float *devPtrA;
    /* float *devPtrB, *devPtrC; */
//includeuse valloc instead of malloc
    /* A = (float*) malloc(4096); */
    checkCudaErrors(hipHostMalloc(&A, 4*4096));
    /* B = (float*) valloc(memsize); */
    /* C = (float*) valloc(memsize); */
    printf("3\n");
    checkCudaErrors(hipHostRegister(A, memsize, hipHostRegisterMapped));
    printf("3-1\n");
    /* hipHostRegister(B, memsize, hipHostRegisterMapped); */
    /* printf("3-2\n"); */
    /* hipHostRegister(C, memsize, hipHostRegisterMapped); */
    /* printf("4\n"); */
    /* for (int i = 0; i < SIZE; i++) { */
    /*         A[i] = B[i] = i; */
    /*     } */

    hipHostGetDevicePointer((void **) &devPtrA, (void *) A, 0);
    /* hipHostGetDevicePointer((void **) &devPtrB, (void *) B, 0); */
    /* hipHostGetDevicePointer((void **) &devPtrC, (void *) C, 0); */

    /* vecAdd<<<1, N>>>(devPtrA, devPtrB, devPtrC); */
    hipDeviceSynchronize();

    /* for (int i = 0; i < SIZE; i++) */
    /*     printf("C[%d]=%f\n", i, C[i]); */

    hipHostUnregister(A);
    /* hipHostUnregister(B); */
    /* hipHostUnregister(C); */
    free(A);
    printf("3-2");
    /* free(B); */
    /* free(C); */

}

