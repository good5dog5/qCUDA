#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void kernel(float* output, float* input)
{

}


int main() {

    int size = 10;

    // Set flag to enable zero copy access
    hipSetDeviceFlags(hipDeviceMapHost);
 
    // Host Arrays
    float* h_in  = NULL;
    float* h_out = NULL;
    
    h_in  = (float *) malloc(size*sizeof(float));
    h_out = (float *) malloc(size*sizeof(float));

    // Device arrays
    float *d_out, *d_in;
    // Get device pointer from host memory. No allocation or memcpy
    hipHostGetDevicePointer((void **)&d_in,  (void *) h_in , 0);
    hipHostGetDevicePointer((void **)&d_out, (void *) h_out, 0);
 
    // Launch the GPU kernel
    kernel<<<1, 1>>>(d_out, d_in);
 
    // No need to copy d_out back
    // Continue processing on host using h_out
    return 0;
}
