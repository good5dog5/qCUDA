#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>

#define N 100

__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}
void dump(float * result)
{
    printf("Result is\n");
    for(int i=0; i<N; i++)
        printf("%f ", result[i]);
    printf("\n");
}

void valid(float *result)
{
    for(int i=0; i<N; i++)
        if (result[i] != 20.0) {
            printf("Failed\n");
            return;
        }
    printf("cudaRegisterFunction PASSED\n");
}
int main(void)
{
    float *x, *y;
    float *x_gpu, *y_gpu;

    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    hipMalloc(&x_gpu, N * sizeof(float));
    hipMalloc(&y_gpu, N * sizeof(float));

    for(int i=0; i<N; i++) {
        x[i] = 5.0;
        y[i] = 10.0;
    }

    hipMemcpy(x_gpu, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, y, N * sizeof(float), hipMemcpyHostToDevice);


    saxpy<<<1024,256>>>(N, 2.0, x_gpu, y_gpu);

    hipMemcpy(x, x_gpu, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, y_gpu, N * sizeof(float), hipMemcpyDeviceToHost);
    /* dump(y); */
    valid(y);

    free(x);
    free(y);
    hipFree(x_gpu);
    hipFree(y_gpu);


}
